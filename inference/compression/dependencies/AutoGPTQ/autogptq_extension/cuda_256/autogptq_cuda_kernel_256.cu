#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// atomicAdd for double-precision floating-point numbers on hardware with
// compute capability < 6.0 from:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
// #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
// __device__ double atomicAdd(
//     double* address,
//     double val
// ) {
//   unsigned long long int* address_as_ull = (unsigned long long int*)address;
//   unsigned long long int old = *address_as_ull, assumed;
//
//   do {
//     assumed = old;
//     old = atomicCAS(
//       address_as_ull,
//       assumed,
//       __double_as_longlong(val + __longlong_as_double(assumed))
//     );
//
//   // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//   } while (assumed != old);
//
//   return __longlong_as_double(old);
// }
// #endif

#if (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700) || defined(USE_ROCM)
// adapted from https://github.com/torch/cutorch/blob/master/lib/THC/THCAtomics.cuh

__device__ __forceinline__ void atomicAdd(c10::Half* address, c10::Half val) {
    unsigned int *address_as_ui = reinterpret_cast<unsigned int *>(reinterpret_cast<char *>(address) - (reinterpret_cast<size_t>(address) & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        unsigned short hsum = reinterpret_cast<size_t>(address) & 2 ? (old >> 16) : (old & 0xffff);
        hsum += val;
        old = reinterpret_cast<size_t>(address) & 2
                 ? (old & 0xffff) | (hsum << 16)
                 : (old & 0xffff0000) | hsum;
        old = atomicCAS(address_as_ui, assumed, old);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
}
__device__ __forceinline__ void atomicAdd(__half* address, c10::Half val) {
    unsigned int * address_as_ui = (unsigned int *) ((char *)address - ((size_t)address & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        __half_raw hsum;
        hsum.x = (size_t)address & 2 ? (old >> 16) : (old & 0xffff);
        half tmpres = __hadd(hsum, val);
        hsum = __half_raw(tmpres);
        old = (size_t)address & 2 ? (old & 0xffff) | (hsum.x << 16) : (old & 0xffff0000) | hsum.x;
        old = atomicCAS(address_as_ui, assumed, old);
    } while (assumed != old);
}
#endif


template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

__global__ void VecQuant2MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const    int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

__global__ void VecQuant3MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const    int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);

__global__ void VecQuant4MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const    int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
);


const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT2 =  16;
const int BLOCKHEIGHT3 =  24;
const int BLOCKHEIGHT4 =  32;
const int BLOCKHEIGHT8 =  64;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}


void vecquant2matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant2matmul_cuda", ([&] {
      VecQuant2MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 16;
  int k;
  unsigned int g;
  scalar_t w_tmp;

  int z_w = w / 16;
  int z_mod = (w % 16) * 2;

  float weight[BLOCKWIDTH];

  for (k = 0; k <  BLOCKWIDTH; ++k){
	int k_w = (k / 16);
	int k_bit = (k % 16) * 2;

    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod & 0x3) + 1);

    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0x3);

	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){
	res = 0;

    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
    __syncthreads();
  }
}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = (h / 3) * 32;
  int k;
  unsigned int g;
  scalar_t w_tmp;

  int z_w = (w / 32) * 3;
  int z_mod = w % 32;
  int z_bit;
  unsigned int z_tmp;
  if (z_mod != 10){
    if (z_mod != 21){
      z_bit = z_mod;
      if (z_bit > 21){
        z_bit -= 22;
        z_bit *= 3;
        z_bit += 2;
        z_w += 2;
      } else if (z_bit > 10){
        z_bit -= 11;
        z_bit *= 3;
        z_bit += 1;
        z_w += 1;
      } else {
        z_bit *= 3;
      }
    } else {
      z_w += 1;
    }
  }

  float weight[BLOCKWIDTH];

  for (k = 0; k <  BLOCKWIDTH; ++k){
	int k_w = (k / 32) * 3;
	int k_mod = k % 32;
	int k_bit;

	if (k_mod != 10){
	  if (k_mod != 21){
        k_bit = k_mod;
        if (k_bit > 21){
		  k_bit -= 22;
		  k_bit *= 3;
		  k_bit += 2;
		  k_w += 2;
        } else if (k_bit > 10){
		  k_bit -= 11;
		  k_bit *= 3;
		  k_bit += 1;
		  k_w += 1;
        } else {
		  k_bit *= 3;
        }
	  } else {
        k_w += 1;
	  }
	}

    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero;
    if (z_mod == 10) {
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 30) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 2) & 0x4);
      zero = scalar_t((z_tmp) + 1);
    } else if (z_mod == 21){
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 31) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 1) & 0x6);
      zero = scalar_t((z_tmp) + 1);
    } else {
      zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_bit) & 0x7) + 1);
    }

    if (k_mod == 10) {
      w_tmp = (as_unsigned(mat[i + (k_w * width)]) >> 30) | ((as_unsigned(mat[i + ((k_w + 1)* width)]) << 2) & 0x4);
    } else if (k_mod == 21){
      w_tmp = (as_unsigned(mat[i + (k_w * width)]) >> 31) | ((as_unsigned(mat[i + ((k_w + 1)* width)]) << 1) & 0x6);
    } else {
      w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0x7);
    }
	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){
	res = 0;

    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
    __syncthreads();
  }
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda", ([&] {
      VecQuant4MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 8;
  int k;
  unsigned int g;
  scalar_t w_tmp;


  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  float weight[BLOCKWIDTH];

  for (k = 0; k <  BLOCKWIDTH; ++k){
	int k_w = (k / 8);
	int k_bit = (k % 8) * 4;

    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1);

    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xF);

	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){
	res = 0;

    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
    __syncthreads();
  }
}

void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 4;
  int k;
  unsigned int g;
  scalar_t w_tmp;

  int z_w = w / 4;
  int z_mod = (w % 4) * 8;

  float weight[BLOCKWIDTH];

  for (k = 0; k <  BLOCKWIDTH; ++k){
	int k_w = (k / 4);
	int k_bit = (k % 4) * 8;

    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xFF) + 1);

    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xFF);

	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){
	res = 0;

    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
    __syncthreads();
  }
}


void vecquant2matmul_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant2matmul_cuda_old", ([&] {
      VecQuant2MatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 16;
  int k = 0;

  int z_w = w / 16;
  int z_mod = (w % 16) * 2;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);

    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod & 0x3) + 1);

    res += (scale * scalar_t((tmp >> 0) & 0x3) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 2) & 0x3) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 4) & 0x3) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 6) & 0x3) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 8) & 0x3) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 10) & 0x3) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 12) & 0x3) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 14) & 0x3) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp >> 16) & 0x3) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp >> 18) & 0x3) - zero) * blockvec[k + 9];
    res += (scale * scalar_t((tmp >> 20) & 0x3) - zero) * blockvec[k + 10];
    res += (scale * scalar_t((tmp >> 22) & 0x3) - zero) * blockvec[k + 11];
    res += (scale * scalar_t((tmp >> 24) & 0x3) - zero) * blockvec[k + 12];
    res += (scale * scalar_t((tmp >> 26) & 0x3) - zero) * blockvec[k + 13];
    res += (scale * scalar_t((tmp >> 28) & 0x3) - zero) * blockvec[k + 14];
    res += (scale * scalar_t((tmp >> 30) & 0x3) - zero) * blockvec[k + 15];

    i += width;
    k += 16;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant3matmul_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda_old", ([&] {
      VecQuant3MatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = (h / 3) * 32;
  int k = 0;

  int z_w = (w / 32) * 3;
  int z_mod = w % 32;
  int z_bit;

  if (z_mod != 10){
    if (z_mod != 21){
      z_bit = z_mod;
      if (z_bit > 21){
        z_bit -= 22;
        z_bit *= 3;
        z_bit += 2;
        z_w += 2;
      } else if (z_bit > 10){
        z_bit -= 11;
        z_bit *= 3;
        z_bit += 1;
        z_w += 1;
      } else {
        z_bit *= 3;
      }
    } else {
      z_w += 1;
    }
  }

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;
  unsigned int z_tmp;

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);

    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero;
    if (z_mod == 10) {
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 30) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 2) & 0x4);
      zero = scale * scalar_t((z_tmp) + 1);
    } else if (z_mod == 21){
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 31) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 1) & 0x6);
      zero = scale * scalar_t((z_tmp) + 1);
    } else {
      zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_bit) & 0x7) + 1);
    }

    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];

    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;

    res += (scale * scalar_t((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];

    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;

    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];

    i += width;
    k += 10;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda_old", ([&] {
      VecQuant4MatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;

  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);

    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1);

    res += (scale * scalar_t((tmp >> 0) & 0xF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 4) & 0xF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 8) & 0xF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 12) & 0xF) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 16) & 0xF) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 20) & 0xF) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 24) & 0xF) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 28) & 0xF) - zero) * blockvec[k + 7];

    i += width;
    k += 8;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant8matmul_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda_old", ([&] {
      VecQuant8MatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 4;
  int k = 0;

  int z_w = w / 4;
  int z_mod = (w % 4) * 8;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);

    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xFF) + 1);

    res += (scale * scalar_t((tmp >> 0) & 0xFF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 8) & 0xFF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 16) & 0xFF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 24) & 0xFF) - zero) * blockvec[k + 3];

    i += width;
    k += 4;
  }

  atomicAdd(&mul[b * width + w], res);
}


void vecquant2matmul_faster_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize,
  int vec_height
) {
  int batch = vec.size(0);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant2MatMulKernelFaster_old<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<int>(),
    batch, vec_height, height, width, zero_width, groupsize
  );
}

__global__ void VecQuant2MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const  	 int* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  const int blockwidth2 = BLOCKWIDTH / 2;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[16][16];
  int val = threadIdx.x / 16;
  int off = threadIdx.x % 16;
  for (; val < 16; val += BLOCKWIDTH / 16) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0x3), __int2half_rn(val >> 2)
    );
  }

  int i = width * h + w;
  int g_h = h * 16;
  int k = 0;

  int z_w = w / 16;
  int z_mod = (w % 16) * 2;

  float res = 0;
  half2 res2;

  unsigned int tmp;

  __syncthreads();

  while (k < blockwidth2) {
    int g = (g_h + (k * 2)) / groupsize;
	float scale_f = scales[g * width + w];
    half2 scale = __float2half2_rn(scale_f);
    half2 zero = __float2half2_rn(-(scale_f * (((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0x3) + 1)));

    std::memset(&res2, 0, sizeof(half2));
    tmp = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  0) & 0xf][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  4) & 0xf][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  8) & 0xf][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 12) & 0xf][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xf][off], scale, zero), blockvec[k + 4], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 20) & 0xf][off], scale, zero), blockvec[k + 5], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xf][off], scale, zero), blockvec[k + 6], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 28) & 0xf][off], scale, zero), blockvec[k + 7], res2);
	i += width;
    k += 8;
    res += __low2float(res2) + __high2float(res2);
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant3matmul_faster_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize,
  int vec_height
) {
  int batch = vec.size(0);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3MatMulKernelFaster_old<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<int>(),
    batch, vec_height, height, width, zero_width, groupsize
  );
}

__global__ void VecQuant3MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const  	 int* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  const int blockwidth2 = BLOCKWIDTH / 2;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[64][32];
  int val = threadIdx.x / 32;
  int off = threadIdx.x % 32;
  for (; val < 64; val += BLOCKWIDTH / 32) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0x7), __int2half_rn(val >> 3)
    );
  }

  int i = width * h + w;
  int g_h = (h / 3) * 32;
  int k = 0;

  int z_w = (w / 32) * 3;
  int z_mod = w % 32;
  int z_bit;

  if (z_mod != 10){
    if (z_mod != 21){
      z_bit = z_mod;
      if (z_bit > 21){
        z_bit -= 22;
        z_bit *= 3;
        z_bit += 2;
        z_w += 2;
      } else if (z_bit > 10){
        z_bit -= 11;
        z_bit *= 3;
        z_bit += 1;
        z_w += 1;
      } else {
        z_bit *= 3;
      }
    } else {
      z_w += 1;
    }
  }

  float res = 0;
  half2 res2;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;
  unsigned int z_tmp;

  __syncthreads();

  while (k < blockwidth2) {
    int g = (g_h + (k * 2)) / groupsize;
	float scale_f = scales[g * width + w];
    half2 scale = __float2half2_rn(scale_f);
    half2 zero;
    if (z_mod == 10) {
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 30) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 2) & 0x4);
      zero = __float2half2_rn(-(scale_f * ((z_tmp) + 1)));
    } else if (z_mod == 21){
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 31) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 1) & 0x6);
      zero = __float2half2_rn(-(scale_f * ((z_tmp) + 1)));
    } else {
      zero = __float2half2_rn(-(scale_f * (((as_unsigned(zeros[g * zero_width + z_w]) >> z_bit) & 0x7) + 1)));
    }

    std::memset(&res2, 0, sizeof(half2));
    tmp1 = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x3c);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 5], res2);
    tmp2 >>= 4;
    k += 6;
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 24) | ((tmp1 << 4) & 0x30);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 4], res2);
    tmp1 >>= 2;
    k += 5;
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    k += 5;
    res += __low2float(res2) + __high2float(res2);
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_faster_cuda_old(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize,
  int vec_height
) {
  int batch = vec.size(0);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernelFaster_old<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<int>(),
    batch, vec_height, height, width, zero_width, groupsize
  );
}

__global__ void VecQuant4MatMulKernelFaster_old(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const  	 int* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  const int blockwidth2 = BLOCKWIDTH / 2;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[256][8];
  int val = threadIdx.x / 8;
  int off = threadIdx.x % 8;
  for (; val < 256; val += BLOCKWIDTH / 8) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0xF), __int2half_rn(val >> 4)
    );
  }

  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;

  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  float res = 0;
  half2 res2;

  unsigned int tmp;

  __syncthreads();

  while (k < blockwidth2) {
    int g = (g_h + (k * 2)) / groupsize;
	float scale_f = scales[g * width + w];

    half2 scale = __float2half2_rn(scale_f);
    half2 zero = __float2half2_rn(-(scale_f * (((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1)));

    //std::memset(&res2, 0, sizeof(half2));

    //res2 = __float2half2_rn((float)0.);

    std::memset(&res2, 0, sizeof(half2));
    tmp = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  0) & 0xff][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  8) & 0xff][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xff][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xff][off], scale, zero), blockvec[k + 3], res2);
	i += width;
    k += 4;

    res += __low2float(res2) + __high2float(res2);

  }

  atomicAdd(&mul[b * width + w], res);
}
